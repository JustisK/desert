#include "hip/hip_runtime.h"
#define THREADS _THREADS_

/*__device__ inline void atomicSaxpy(float *adr,*/
/*                            const float ia,*/
/*                            const float k){*/
/*  float val;*/
/*  while (true){*/
/*    val = atomicExch(adr, -1.0f);*/
/*    if (val > -1.0f){*/
/*      val = ia*val + k;*/
/*      atomicExch(adr, val);*/
/*      break;*/
/*    }*/
/*  }*/
/*}*/

/*__device__ inline void atomicSaxpy(float *adr,*/
/*                            const float ia,*/
/*                            const float k){*/
/*  float old = -1.0f;*/
/*  float new_old;*/

/*  do {*/
/*    new_old = atomicExch(adr, -1.0f);*/
/*    new_old = ia*new_old + k;*/
/*  } while ((old = atomicExch(adr, new_old)) != -1.0f);*/
/*}*/

__device__ inline void atomicSaxpy(float *adr,
                            const float ia,
                            const float k){

  float old = atomicExch(adr, -1.0f);
  float new_old = old*ia + k;

  while ((old = atomicExch(adr, new_old)) != -1.0f) {
    new_old = atomicExch(adr, -1.0f);
    new_old = old*ia +k;
  }
}


__global__ void dot(const int n,
                    const int imsize,
                    float *img,
                    const float *xy,
                    const float *rgba){
  const int i = blockIdx.x*THREADS + threadIdx.x;

  if (i >= n){
    return;
  }

  const int ii = 2*i;
  const int x = (int)floor(xy[ii]* (float)imsize);
  const int y = (int)floor(xy[ii+1]* (float)imsize);

  if (x>=imsize || x<0 || y>=imsize || y<0){
    return;
  }

  const float a = rgba[3];
  const float ia = 1.0-a;

  const int ij = 4*(x*imsize+y);

  atomicSaxpy(&img[ij], ia, rgba[0]);
  atomicSaxpy(&img[ij+1], ia, rgba[1]);
  atomicSaxpy(&img[ij+2], ia, rgba[2]);
  atomicSaxpy(&img[ij+3], ia, rgba[3]);
}

